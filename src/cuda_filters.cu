#include "hip/hip_runtime.h"
#include "utils.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#define SOBEL_R 1

#define TILE_HEIGHT 16
#define TILE_WIDTH 16

#define BLOCK_WIDTH (TILE_WIDTH + (2 * SOBEL_R))
#define BLOCK_HEIGHT (TILE_HEIGHT + (2 * SOBEL_R))

__global__ void gray_filter_kernel(pixel *p, unsigned size)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size)
  {
    int moy = (p[i].r + p[i].g + p[i].b) / 3;
    moy = max(0, min(255, moy));

    p[i].r = moy;
    p[i].g = moy;
    p[i].b = moy;
  }
}

// Inspired by Nvidia CUDA samples
__global__ void sobel_filter_kernel(pixel *p, pixel *new_p, int width, int height)
{
  __shared__ int smem[BLOCK_HEIGHT * BLOCK_WIDTH];

  int x = blockIdx.x * TILE_WIDTH + threadIdx.x - SOBEL_R;
  int y = blockIdx.y * TILE_HEIGHT + threadIdx.y - SOBEL_R;

  unsigned i = y * width + x;
  unsigned smem_i = threadIdx.y * BLOCK_WIDTH + threadIdx.x;

  if (x >= 0 && x < width && y >= 0 && y < height)
  {
    smem[smem_i] = p[i].b;
  }

  __syncthreads();

  if (threadIdx.x >= SOBEL_R && threadIdx.x < TILE_WIDTH + SOBEL_R &&
      threadIdx.y >= SOBEL_R && threadIdx.y < TILE_HEIGHT + SOBEL_R &&
      x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
  {
    float delta_x = 2 * (smem[smem_i + 1] - smem[smem_i - 1]) +
                    (smem[smem_i - BLOCK_WIDTH + 1] - smem[smem_i - BLOCK_WIDTH - 1]) +
                    (smem[smem_i + BLOCK_WIDTH + 1] - smem[smem_i + BLOCK_WIDTH - 1]);

    float delta_y = 2 * (smem[smem_i + BLOCK_WIDTH] - smem[smem_i - BLOCK_WIDTH]) +
                    (smem[smem_i + BLOCK_WIDTH - 1] - smem[smem_i - BLOCK_WIDTH - 1]) +
                    (smem[smem_i + BLOCK_WIDTH + 1] - smem[smem_i - BLOCK_WIDTH + 1]);

    int new_val = sqrt(delta_x * delta_x + delta_y * delta_y) / 4;

    new_p[i].r = (new_val > 50) * 255;
    new_p[i].g = (new_val > 50) * 255;
    new_p[i].b = (new_val > 50) * 255;
  }
}

__global__ void horizontal_pass(pixel *p, pixel *p_out, int width, int height, int size)
{
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx >= height)
  {
    return;
  }

  int sum = 0;
  int idx = thread_idx * width;
  for (int i = 0; i < 2 * size + 1; i++)
  {
    sum += p[idx + i].r;
  }

  for (int i = 0; i < width - 2 * size; i++)
  {
    p_out[idx + size + i].r = sum;
    p_out[idx + size + i].g = sum;
    p_out[idx + size + i].b = sum;

    sum += p[idx + i + 2 * size + 1].r;
    sum -= p[idx + i].r;
  }
}

__global__ void vertical_pass(pixel *p, pixel *p_out, int width, int height, int size)
{
  int sum = 0;
  int idx = blockIdx.x * blockDim.x + threadIdx.x + size;

  if (idx >= width - size)
  {
    return;
  }

  for (int i = 0; i < 2 * size + 1; i++)
  {
    sum += p[idx + width * i].r;
  }

  for (int i = 0; i < height - 2 * size; i++)
  {
    p_out[idx + width * (i + size)].r = sum;
    p_out[idx + width * (i + size)].g = sum;
    p_out[idx + width * (i + size)].b = sum;

    sum += p[idx + width * (i + 2 * size + 1)].r;
    sum -= p[idx + width * i].r;
  }
}

__global__ void normalize_pixel_values(pixel *img, int width, int height, int size, int area)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width - size || y >= height - size || x < size || y < size)
  {
    return;
  }

  int idx = y * width + x;
  img[idx].r /= area;
  img[idx].g /= area;
  img[idx].b /= area;
}

extern "C" void cuda_apply_gray_filter_once(img *image_d)
{
  const size_t block_size = 256;
  const size_t num_blocks = (image_d->width * image_d->height + block_size - 1) / block_size;
  gray_filter_kernel<<<num_blocks, block_size>>>(image_d->p, image_d->width * image_d->height);
}

extern "C" void cuda_apply_blur_filter_once(img *image, int size, int threshold)
{
  pixel *new_p_d = nullptr;
  hipMalloc(&new_p_d, image->width * image->height * sizeof(pixel));
  hipMemcpy(new_p_d, image->p, image->width * image->height * sizeof(pixel), hipMemcpyDeviceToDevice);

  // second dimension is used to blur either the bottom or top of the image
  const dim3 block_size(256, 1);
  const dim3 num_hor_blocks((image->height + block_size.x - 1) / block_size.x, 1);
  const dim3 num_vert_blocks((image->width + block_size.x - 1) / block_size.x, 1);

  const dim3 block_size_norm(BLOCK_WIDTH, BLOCK_HEIGHT);
  const dim3 num_norm_blocks((image->width + block_size_norm.x - 1) / block_size_norm.x, (image->height + block_size_norm.y - 1) / block_size_norm.y);

  horizontal_pass<<<num_hor_blocks, block_size>>>(image->p, new_p_d, image->width, image->height / 10, size);
  vertical_pass<<<num_vert_blocks, block_size>>>(new_p_d, image->p, image->width, image->height / 10, size);
  normalize_pixel_values<<<num_norm_blocks, block_size_norm>>>(image->p, image->width, image->height / 10, size, (2 * size + 1) * (2 * size + 1));

  // TODO: implement this using block y dimension
  int offset = image->width * (image->height - image->height / 10 - 1);
  horizontal_pass<<<num_hor_blocks, block_size>>>(image->p + offset, new_p_d + offset, image->width, image->height / 10, size);
  vertical_pass<<<num_vert_blocks, block_size>>>(new_p_d + offset, image->p + offset, image->width, image->height / 10, size);
  normalize_pixel_values<<<num_norm_blocks, block_size_norm>>>(image->p + offset, image->width, image->height / 10, size, (2 * size + 1) * (2 * size + 1));

  hipFree(new_p_d);
}

extern "C" void cuda_apply_sobel_filter_once(img *image)
{
  pixel *new_p_d = nullptr;
  hipMalloc(&new_p_d, image->width * image->height * sizeof(pixel));
  // TODO: Fix, this works but is not efficient. I tried doing it in the kernel but it didn't work
  hipMemcpy(new_p_d, image->p, image->width * image->height * sizeof(pixel), hipMemcpyDeviceToDevice);

  const dim3 block_size(BLOCK_WIDTH, BLOCK_HEIGHT);
  const dim3 num_blocks((image->width + TILE_WIDTH - 1) / TILE_WIDTH, (image->height + TILE_HEIGHT - 1) / TILE_HEIGHT);
  sobel_filter_kernel<<<num_blocks, block_size>>>(image->p, new_p_d, image->width, image->height);

  hipFree(image->p);
  image->p = new_p_d;
}

extern "C" void cuda_pipe(img *image)
{
  /* Allocate memory for the image on device*/
  img image_d = *image;
  hipMalloc(&image_d.p, image_d.width * image_d.height * sizeof(pixel));
  hipMemcpy(image_d.p, image->p, image->width * image->height * sizeof(pixel), hipMemcpyHostToDevice);

  /* Convert the pixels into grayscale */
  cuda_apply_gray_filter_once(&image_d);

  /* Apply blur filter with convergence value */
  cuda_apply_blur_filter_once(&image_d, 5, 20);

  /* Apply sobel filter on pixels */
  // cuda_apply_sobel_filter_once(&image_d);

  /* Copy the pixels back to the host and frees memmory */
  hipDeviceSynchronize();
  hipMemcpy(image->p, image_d.p, image->width * image->height * sizeof(pixel), hipMemcpyDeviceToHost);
  hipFree(image_d.p);
}