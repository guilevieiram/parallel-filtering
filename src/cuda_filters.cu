#include "hip/hip_runtime.h"
#include "utils.h"

#include <hip/hip_runtime.h>

#define SOBEL_R 1

#define TILE_HEIGHT 16
#define TILE_WIDTH 16

#define BLOCK_WIDTH (TILE_WIDTH + (2 * SOBEL_R))
#define BLOCK_HEIGHT (TILE_HEIGHT + (2 * SOBEL_R))

__global__ void gray_filter_kernel(pixel *p, unsigned size)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size)
  {
    int moy = p[i].r + p[i].g + p[i].b;
    moy = 255 * (moy > 3 * 255);

    p[i].r = moy;
    p[i].g = moy;
    p[i].b = moy;
  }
}

__global__ void blur_filter_kernel(pixel *p, pixel *new_p, int width, int height, int size, int threshold)
{
}

// Inspired by Nvidia CUDA samples
__global__ void sobel_filter_kernel(pixel *p, pixel *new_p, int width, int height)
{
  __shared__ int smem[BLOCK_HEIGHT * BLOCK_WIDTH];

  int x = blockIdx.x * TILE_WIDTH + threadIdx.x - SOBEL_R;
  int y = blockIdx.y * TILE_HEIGHT + threadIdx.y - SOBEL_R;

  unsigned i = y * width + x;
  unsigned smem_i = threadIdx.y * BLOCK_WIDTH + threadIdx.x;

  if (x >= 0 && x < width && y >= 0 && y < height)
  {
    smem[smem_i] = p[i].b;
  }

  __syncthreads();

  if (threadIdx.x >= SOBEL_R && threadIdx.x < TILE_WIDTH + SOBEL_R &&
      threadIdx.y >= SOBEL_R && threadIdx.y < TILE_HEIGHT + SOBEL_R &&
      x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
  {
    float delta_x = 2 * (smem[smem_i + 1] - smem[smem_i - 1]) +
                    (smem[smem_i - BLOCK_WIDTH + 1] - smem[smem_i - BLOCK_WIDTH - 1]) +
                    (smem[smem_i + BLOCK_WIDTH + 1] - smem[smem_i + BLOCK_WIDTH - 1]);

    float delta_y = 2 * (smem[smem_i + BLOCK_WIDTH] - smem[smem_i - BLOCK_WIDTH]) +
                    (smem[smem_i + BLOCK_WIDTH - 1] - smem[smem_i - BLOCK_WIDTH - 1]) +
                    (smem[smem_i + BLOCK_WIDTH + 1] - smem[smem_i - BLOCK_WIDTH + 1]);

    int new_val = sqrt(delta_x * delta_x + delta_y * delta_y) / 4;

    new_p[i].r = (new_val > 50) * 255;
    new_p[i].g = (new_val > 50) * 255;
    new_p[i].b = (new_val > 50) * 255;
  }
}

extern "C" void cuda_apply_gray_filter_once(img *image_d)
{
  const size_t block_size = 256;
  const size_t num_blocks = (image_d->width * image_d->height + block_size - 1) / block_size;
  gray_filter_kernel<<<num_blocks, block_size>>>(image_d->p, image_d->width * image_d->height);
}

extern "C" void cuda_apply_blur_filter_once(img *image, int size, int threshold)
{
}

extern "C" void cuda_apply_sobel_filter_once(img *image)
{
  pixel *new_p_d = nullptr;
  hipMalloc(&new_p_d, image->width * image->height * sizeof(pixel));

  const dim3 block_size(BLOCK_WIDTH, BLOCK_HEIGHT);
  const dim3 num_blocks((image->width + TILE_WIDTH - 1) / TILE_WIDTH, (image->height + TILE_HEIGHT - 1) / TILE_HEIGHT);
  sobel_filter_kernel<<<num_blocks, block_size>>>(image->p, new_p_d, image->width, image->height);

  hipFree(image->p);
  image->p = new_p_d;  
}

extern "C" void cuda_pipe(img *image)
{
  /* Allocate memory for the image on device*/
  img image_d = *image;
  hipMalloc(&image_d.p, image_d.width * image_d.height * sizeof(pixel));
  hipMemcpy(image_d.p, image->p, image->width * image->height * sizeof(pixel), hipMemcpyHostToDevice);

  /* Convert the pixels into grayscale */
  cuda_apply_gray_filter_once(&image_d);

  /* Apply blur filter with convergence value */
  cuda_apply_blur_filter_once(&image_d, 5, 20);

  /* Apply sobel filter on pixels */
  cuda_apply_sobel_filter_once(&image_d);

  /* Copy the pixels back to the host and frees memmory */
  hipDeviceSynchronize();
  hipMemcpy(image->p, image_d.p, image->width * image->height * sizeof(pixel), hipMemcpyDeviceToHost);
  hipFree(image_d.p);
}